#include "hip/hip_runtime.h"
/*
 * Particle System with OpenGL using the GPU for physics calculations
 * (c) 2023 by Jessica Leyba
 * softwave.com
*/

#include <iostream>
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <glm/glm.hpp> // vec3, vec4, ivec4, mat4
#include <glm/gtc/matrix_transform.hpp>                       
#include <glm/gtc/type_ptr.hpp> // value_ptr
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h> // offsetof
#include <vector>

// Variables 
const int SCREEN_WIDTH = 1600;
const int SCREEN_HEIGHT = 900;
const int PARTICLE_COUNT = 25000;
// Simulation constants
const float GRAVITY = -0.1f;


struct Particle
{
    glm::vec3 position;
    glm::vec3 velocity;
    glm::vec3 color;
    float mass;
};

Particle *particles;

// CUDA kernel for calculating particle positions 
__global__ void updateParticles(Particle* particles, int n)
{
const float G = 6.67430e-07; 
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
    {
        float Fgx = 0.0f, Fgy = 0.0f, Fgz = 0.0f;

        for (int j = 0; j < n; j++)
        {
            if (index != j)
            {
                float dx = particles[j].position.x - particles[index].position.x;
                float dy = particles[j].position.y - particles[index].position.y;
                float dz = particles[j].position.z - particles[index].position.z;

                float distSqr = dx*dx + dy*dy + dz*dz;
                 // Add softening to avoid infinities
                float dist = sqrt(distSqr) + 0.0051f;

                float force = G / distSqr;

                Fgx += force * dx / dist;
                Fgy += force * dy / dist;
                Fgz += force * dz / dist;
            }
        }

        particles[index].velocity.x += Fgx * particles[index].mass;
        particles[index].velocity.y += Fgy * particles[index].mass;
        particles[index].velocity.z += Fgz * particles[index].mass;

        particles[index].position.x += particles[index].velocity.x;
        particles[index].position.y += particles[index].velocity.y;
        particles[index].position.z += particles[index].velocity.z;

        // Add some spin like a galaxy
        particles[index].velocity.x += particles[index].position.y * 0.00008f;
        particles[index].velocity.y -= particles[index].position.x * 0.00008f;
        

        // Set color based on position
        particles[index].color.x = (particles[index].position.x + 2.0f) / 2.0f;
        particles[index].color.y = (particles[index].position.y + 1.0f) / 2.0f;
        particles[index].color.z = (particles[index].position.z + 2.0f) / 2.0f;

        //Add to color based on the inverse of the distance from the center
        float dist = sqrt(particles[index].position.x * particles[index].position.x + particles[index].position.y * particles[index].position.y + particles[index].position.z * particles[index].position.z);
        particles[index].color.x += 1.0f / dist;
        particles[index].color.y += 1.0f / dist;
        particles[index].color.z += 1.0f / dist;
    }
}


char* loadShaderSource(const char* filepath)
{
    FILE* file = fopen(filepath, "rb");
    if (!file)
    {
        fprintf(stderr, "Error: Could not open shader file %s\n", filepath);
        return NULL;
    }

    // Go to the end of the file to determine its size
    fseek(file, 0, SEEK_END);
    long filesize = ftell(file);
    fseek(file, 0, SEEK_SET);

    // Allocate buffer for the source code and read it in
    char* buffer = (char*)malloc(filesize + 1);  // +1 for null terminator
    if (!buffer)
    {
        fprintf(stderr, "Error: Could not allocate memory for shader source\n");
        fclose(file);
        return NULL;
    }
    
    fread(buffer, 1, filesize, file);
    buffer[filesize] = '\0';  // Null-terminate the buffer

    fclose(file);
    return buffer;
}


int main(void)
{
    // Initialise GLFW
    if (!glfwInit())
    {
        fprintf(stderr, "Error: Could not initialise GLFW\n");
        return -1;
    }

    // Open a window and create its OpenGL context
    GLFWwindow* window = glfwCreateWindow(SCREEN_WIDTH, SCREEN_HEIGHT, "Big Bang", NULL, NULL);
    if (!window)
    {
        fprintf(stderr, "Error: Could not open window\n");
        glfwTerminate();
        return -1;
    }

    glfwMakeContextCurrent(window);
    // Init GLEW
    glewExperimental = GL_TRUE;
    if (glewInit() != GLEW_OK)
    {
        fprintf(stderr, "Error: Could not initialise GLEW\n");
        glfwTerminate();
        return -1;
    }

    particles = new Particle[PARTICLE_COUNT];
    // Setup simulation
    



    
    // Load and compile shaders
    char* vertexShaderSource = loadShaderSource("vert.glsl");
    char* fragmentShaderSource = loadShaderSource("frag.glsl");
    char* computeShaderSource = loadShaderSource("compute.glsl");

    GLuint vertexShader = glCreateShader(GL_VERTEX_SHADER);
    GLuint fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
    //GLuint computeShader = glCreateShader(GL_COMPUTE_SHADER);

    glShaderSource(vertexShader, 1, &vertexShaderSource, NULL);
    glShaderSource(fragmentShader, 1, &fragmentShaderSource, NULL);
    //glShaderSource(computeShader, 1, &computeShaderSource, NULL);

    GLint success;
    GLchar infoLog[512];
    glCompileShader(vertexShader);
    glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
    if(!success)
    {
        glGetShaderInfoLog(vertexShader, 512, NULL, infoLog);
        std::cerr << "Vertex Shader Compilation Failed\n" << infoLog << std::endl;
    }
    
    glCompileShader(fragmentShader);
    glGetShaderiv(fragmentShader, GL_COMPILE_STATUS, &success);
    if(!success) {
        glGetShaderInfoLog(fragmentShader, 512, NULL, infoLog);
        std::cerr << "Fragment Shader Compilation Failed\n" << infoLog << std::endl;
    }

    GLuint shaderProgram = glCreateProgram();
    glAttachShader(shaderProgram, vertexShader);
    glAttachShader(shaderProgram, fragmentShader);
    glLinkProgram(shaderProgram);
    glUseProgram(shaderProgram);

    // Initialise particle data
    for (int i = 0; i < PARTICLE_COUNT; i++)
    {
        // Position in a sphere
        float theta = (float)rand() / RAND_MAX * 2 * M_PI;
        float phi = (float)rand() / RAND_MAX * 2 * M_PI;
        float r = (float)rand() / RAND_MAX * 0.5f + 0.5f;
        particles[i].position.x = r * sin(theta) * cos(phi);
        particles[i].position.y = r * sin(theta) * sin(phi);
        particles[i].position.z = r * cos(theta); 

        // Random mass between 0.5 and 1.5
        particles[i].mass = (float)rand() / RAND_MAX + 0.5f;
    }

    // Buffer particle data to GPU
    GLuint particleBuffer;
    glGenBuffers(1, &particleBuffer);
    glBindBuffer(GL_SHADER_STORAGE_BUFFER, particleBuffer);
    glBufferData(GL_SHADER_STORAGE_BUFFER, PARTICLE_COUNT * sizeof(Particle), particles, GL_DYNAMIC_DRAW);

    
    // Setup Rendering Params (VAOs, VBOs, etc)
    GLuint particleVBO;
    glGenBuffers(1, &particleVBO);
    glBindBuffer(GL_ARRAY_BUFFER, particleVBO);
    glBufferData(GL_ARRAY_BUFFER, PARTICLE_COUNT * sizeof(Particle), particles, GL_DYNAMIC_DRAW);
    GLuint particleVAO;
    glGenVertexArrays(1, &particleVAO);
    glBindVertexArray(particleVAO);
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(Particle), (GLvoid*)offsetof(Particle, position));
    glEnableVertexAttribArray(0);
    glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, sizeof(Particle), (GLvoid*)offsetof(Particle, velocity));
    glEnableVertexAttribArray(1);
    // Color
    glVertexAttribPointer(2, 3, GL_FLOAT, GL_FALSE, sizeof(Particle), (GLvoid*)offsetof(Particle, color));
    glEnableVertexAttribArray(2);
  
    glUseProgram(shaderProgram);

    // Projection and model and view matrices
    glm::mat4 projectionMatrix = glm::perspective(45.0f, (float)SCREEN_WIDTH / (float)SCREEN_HEIGHT, 0.1f, 100.0f);
    glm::vec3 cameraPos = glm::vec3(0.0f, 0.0f, 3.0f); // Position of the camera in the world
    glm::vec3 cameraTarget = glm::vec3(0.0f, 0.0f, 0.0f); // The point in the world the camera is looking at
    glm::vec3 upVector = glm::vec3(0.0f, 1.0f, 0.0f); 
    glm::mat4 viewMatrix = glm::lookAt(cameraPos, cameraTarget, upVector);
    glm::mat4 modelMatrix = glm::mat4(1.0f);
    // Combined model view projection matrix
    glm::mat4 mvpMatrix = projectionMatrix * viewMatrix * modelMatrix;
    
    GLint uMVPMatrixLocation = glGetUniformLocation(shaderProgram, "uMVPMatrix");
    glUniformMatrix4fv(uMVPMatrixLocation, 1, GL_FALSE, glm::value_ptr(mvpMatrix));
    glBindVertexArray(particleVAO);

    // CUDA setup
    Particle* d_particles;
    size_t size = PARTICLE_COUNT * sizeof(Particle);
    hipError_t err;
    hipMalloc(&d_particles, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Could not allocate memory on GPU\n");
        return -1;
    }

    // Delta time
    float dt = 0.0f;
    
    // Draw
    while (!glfwWindowShouldClose(window))
    {
        // Clear the screen
        glClear(GL_COLOR_BUFFER_BIT);
        

        // Draw particles
        glDrawArrays(GL_POINTS, 0, PARTICLE_COUNT);

        // Copy data from CPU to GPU
        hipMemcpy(d_particles, particles, size, hipMemcpyHostToDevice);

        
        dt = glfwGetTime();
        // Launch CUDA kernel to update particles
        int threadsPerBlock = 256;
        int blocksPerGrid = (PARTICLE_COUNT + threadsPerBlock - 1) / threadsPerBlock;
        updateParticles<<<blocksPerGrid, threadsPerBlock>>>(d_particles, PARTICLE_COUNT);
        // Wait for kernel to finish
        hipDeviceSynchronize();
        // Copy data back from GPU to CPU
        hipMemcpy(particles, d_particles, size, hipMemcpyDeviceToHost);
        // Free GPU memory

        // Update GPU Buffer
        glBindBuffer(GL_ARRAY_BUFFER, particleVBO);
        glBufferSubData(GL_ARRAY_BUFFER, 0, PARTICLE_COUNT * sizeof(Particle), particles);

        glPointSize(4.0f);
        glDrawArrays(GL_POINTS, 0, PARTICLE_COUNT);

        // Swap buffers
        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    // Cleanup
    hipFree(d_particles);
    

    
    return 0;
}



